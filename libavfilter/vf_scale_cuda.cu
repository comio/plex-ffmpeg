
#include <hip/hip_runtime.h>
/*
 * Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

typedef unsigned char uchar;
typedef unsigned short ushort;

#define SHIFTDOWN(val) (dstbase)(val >> abs(2 + shift))
#define SHIFTUP(val)   (dstbase)(val << abs(-shift - 2))

template<class SRC, class DST, int shift, int dither> struct add_conv_shift1_d
{
    typedef DST dstbase;

    __inline__ __device__ DST operator()(SRC i1, SRC i2, SRC i3, SRC i4, ushort d)
    {
        unsigned ret = (unsigned)i1 + (unsigned)i2 + (unsigned)i3 + (unsigned)i4 + ((1 + d) >> (sizeof(SRC) * 8 - dither + 3));

        if (shift > -2)
            return SHIFTDOWN(ret);
        else
            return SHIFTUP(ret);
    }
};

template<class SRC, class DST, int shift, int dither> struct add_conv_shift1
{
    typedef DST dstbase;

    __inline__ __device__ DST operator()(SRC i1, SRC i2, SRC i3, SRC i4, ushort d)
    {
        unsigned ret = (unsigned)i1 + (unsigned)i2 + (unsigned)i3 + (unsigned)i4 + 2;

        if (shift > -2)
            return SHIFTDOWN(ret);
        else
            return SHIFTUP(ret);
    }
};

template<class SRC, class DST, int shift, int dither> struct add_conv_shift2
{
    typedef decltype(DST::x) dstbase;

    __inline__ __device__ DST operator()(SRC i1, SRC i2, SRC i3, SRC i4, ushort d)
    {
        unsigned retx = (unsigned)i1.x + (unsigned)i2.x + (unsigned)i3.x + (unsigned)i4.x + 2;
        unsigned rety = (unsigned)i1.y + (unsigned)i2.y + (unsigned)i3.y + (unsigned)i4.y + 2;

        if (shift > -2)
            return { SHIFTDOWN(retx), SHIFTDOWN(rety) };
        else
            return { SHIFTUP(retx),   SHIFTUP(rety)   };
    }
};

template<class SRC, class DST, int shift, int dither> struct add_conv_shift2_x
{
    __inline__ __device__ DST operator()(SRC i1, SRC i2, SRC i3, SRC i4, ushort d)
    {
        return add_conv_shift1<unsigned, DST, shift, dither>()(i1.x, i2.x, i3.x, i4.x, d);
    }
};

template<class SRC, class DST, int shift, int dither> struct add_conv_shift2_y
{
    __inline__ __device__ DST operator()(SRC i1, SRC i2, SRC i3, SRC i4, ushort d)
    {
        return add_conv_shift1<unsigned, DST, shift, dither>()(i1.y, i2.y, i3.y, i4.y, d);
    }
};

template<class SRC, class DST, int shift, int dither> struct add_conv_shift3
{
    typedef decltype(DST::x) dstbase;

    __inline__ __device__ DST operator()(SRC i1, SRC i2, SRC i3, SRC i4, ushort d)
    {
        unsigned retx = (unsigned)i1.x + (unsigned)i2.x + (unsigned)i3.x + (unsigned)i4.x + 2;
        unsigned rety = (unsigned)i1.y + (unsigned)i2.y + (unsigned)i3.y + (unsigned)i4.y + 2;
        unsigned retz = (unsigned)i1.z + (unsigned)i2.z + (unsigned)i3.z + (unsigned)i4.z + 2;

        if (shift > -2)
            return { SHIFTDOWN(retx), SHIFTDOWN(rety), SHIFTDOWN(retz) };
        else
            return { SHIFTUP(retx),   SHIFTUP(rety),   SHIFTUP(retz)   };
    }
};

template<class SRC, class DST, int shift, int dither> struct add_conv_shift4
{
    typedef decltype(DST::x) dstbase;

    __inline__ __device__ DST operator()(SRC i1, SRC i2, SRC i3, SRC i4, ushort d)
    {
        unsigned retx = (unsigned)i1.x + (unsigned)i2.x + (unsigned)i3.x + (unsigned)i4.x + 2;
        unsigned rety = (unsigned)i1.y + (unsigned)i2.y + (unsigned)i3.y + (unsigned)i4.y + 2;
        unsigned retz = (unsigned)i1.z + (unsigned)i2.z + (unsigned)i3.z + (unsigned)i4.z + 2;
        unsigned retw = (unsigned)i1.w + (unsigned)i2.w + (unsigned)i3.w + (unsigned)i4.w + 2;

        if (shift > -2)
            return { SHIFTDOWN(retx), SHIFTDOWN(rety), SHIFTDOWN(retz), SHIFTDOWN(retw) };
        else
            return { SHIFTUP(retx),   SHIFTUP(rety),   SHIFTUP(retz),   SHIFTUP(retw)   };
    }
};

template<class SRC, class DST, template<class, class, int, int> class conv, int pitch, int shift, int dither>
__inline__ __device__ void Subsample_Bilinear(hipTextureObject_t tex,
                                   DST *dst,
                                   int dst_width, int dst_height, int dst_pitch,
                                   int src_width, int src_height,
                                   hipTextureObject_t ditherTex)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float hscale = (float)src_width / (float)dst_width;
        float vscale = (float)src_height / (float)dst_height;
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;
        // 3-tap filter weights are {wh,1.0,wh} and {wv,1.0,wv}
        float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
        float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
        // Convert weights to two bilinear weights -> {wh,1.0,wh} -> {wh,0.5,0} + {0,0.5,wh}
        float dx = wh / (0.5f + wh);
        float dy = wv / (0.5f + wv);

        SRC i0 = tex2D<SRC>(tex, xi-dx, yi-dy);
        SRC i1 = tex2D<SRC>(tex, xi+dx, yi-dy);
        SRC i2 = tex2D<SRC>(tex, xi-dx, yi+dy);
        SRC i3 = tex2D<SRC>(tex, xi+dx, yi+dy);

        ushort ditherVal = dither ? tex2D<ushort>(ditherTex, xo, yo) : 0;

        dst[yo*(dst_pitch / sizeof(DST))+xo*pitch] = conv<SRC, DST, shift, dither>()(i0, i1, i2, i3, ditherVal);
    }
}

extern "C" {

#define VARIANT(SRC, DST, CONV, SHIFT, PITCH, DITHER, NAME) \
__global__ void Subsample_Bilinear_ ## NAME(hipTextureObject_t tex, \
                                    DST *dst, \
                                    int dst_width, int dst_height, int dst_pitch, \
                                    int src_width, int src_height, \
                                    hipTextureObject_t ditherTex) \
{ \
    Subsample_Bilinear<SRC, DST, CONV, PITCH, SHIFT, DITHER>(tex, dst, dst_width, dst_height, dst_pitch, \
                                                             src_width, src_height, ditherTex); \
}

#define VARIANTSET2(SRC, DST, SHIFT, NAME) \
    VARIANT(SRC,      DST,      add_conv_shift1_d, SHIFT, 1, (sizeof(DST) < sizeof(SRC)) ? sizeof(DST) : 0, NAME) \
    VARIANT(SRC,      DST,      add_conv_shift1,   SHIFT, 1, 0, NAME ## _c) \
    VARIANT(SRC,      DST,      add_conv_shift1,   SHIFT, 2, 0, NAME ## _p2) \
    VARIANT(SRC ## 2, DST ## 2, add_conv_shift2,   SHIFT, 1, 0, NAME ## _2) \
    VARIANT(SRC ## 2, DST,      add_conv_shift2_x, SHIFT, 1, 0, NAME ## _2_u) \
    VARIANT(SRC ## 2, DST,      add_conv_shift2_y, SHIFT, 1, 0, NAME ## _2_v) \
    VARIANT(SRC ## 4, DST ## 4, add_conv_shift4,   SHIFT, 1, 0, NAME ## _4)

#define VARIANTSET(SRC, DST, SRCSIZE, DSTSIZE) \
    VARIANTSET2(SRC, DST, (SRCSIZE - DSTSIZE), SRCSIZE ## _ ## DSTSIZE)

// Straight no-conversion
VARIANTSET(uchar,  uchar,  8,  8)
VARIANTSET(ushort, ushort, 16, 16)

// Conversion between 8- and 16-bit
VARIANTSET(uchar,  ushort, 8,  16)
VARIANTSET(ushort, uchar,  16, 8)

}
